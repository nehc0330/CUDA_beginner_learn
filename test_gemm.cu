#include <cstdio>
#include <hip/hip_runtime.h>
#include "utils.cuh"
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>
int num = 2048 * 8;
// C = A * B

int main()
{
    // sizeof A B C
    
    int M = num;
    int K = num;
    int N = num;

    // print gpu info
    hipDeviceProp_t deviceProp;
    int devID = 0;
    checkCudaErrors(hipSetDevice(devID));
    auto error = hipGetDeviceProperties(&deviceProp, devID);
    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error,
               __LINE__);
        exit(EXIT_FAILURE);
    }
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", devID,
           deviceProp.name, deviceProp.major, deviceProp.minor);

    // ��ʱ
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float elapsed_time = 0.0f;

    // cublas ����
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1.0f, beta = 0.0f;

    // sizeof A B C mem
    const size_t A_mem_size = M * K * sizeof(float);
    const size_t B_mem_size = N * K * sizeof(float);
    const size_t C_mem_size = M * N * sizeof(float);

    float *h_A = (float *)malloc(A_mem_size);
    float *h_B = (float *)malloc(B_mem_size);
    float *h_C_cublas = (float *)malloc(C_mem_size);
    float *h_C_cpu = (float *)malloc(C_mem_size);
    float *h_C_gpu = (float *)malloc(C_mem_size);

    init_matrix(M, K, h_A);
    init_matrix(K, N, h_B);
    memset(h_C_cublas, 0, C_mem_size);
    memset(h_C_cpu, 0, C_mem_size);
    memset(h_C_gpu, 0, C_mem_size);

    float *d_A, *d_B, *d_C, *d_C_cublas;
    checkCudaErrors(hipMalloc((void **)&d_A, A_mem_size));
    checkCudaErrors(hipMalloc((void **)&d_B, B_mem_size));
    checkCudaErrors(hipMalloc((void **)&d_C, C_mem_size));
    checkCudaErrors(hipMalloc((void **)&d_C_cublas, C_mem_size));

    // host2dev
    checkCudaErrors(hipMemcpy(d_A, h_A, A_mem_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, B_mem_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_C_cublas, h_C_cublas, C_mem_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_C, h_C_gpu, C_mem_size, hipMemcpyHostToDevice));
    //-------------------------------------gpu_calc----------------------------------------------------
    checkCudaErrors(hipEventRecord(start, 0)); // 0 Ĭ����
    // test_v1;

    // GlobalMemory(M,K,N,d_A,d_B,d_C);

    // test_v2;

    // ShareMemory(M,K,N,d_A,d_B,d_C);

    // test_v3 21.7ms

    // STRIDE_ShareMemory(M,K,N,d_A,d_B,d_C);

    // test_v4 7.4ms
    // Float4_ShareMemory(M,K,N,d_A,d_B,d_C);

    // test_v5 4.9ms
    // RMem_Float4_ShareMemory(M,K,N,d_A,d_B,d_C);

    // test_v6 4.4ms
    // Transpose_RMem_Float4_ShareMemory(M,K,N,d_A,d_B,d_C);

    // test_v7; 2.9ms
    Buffer_Transpose_RMem_Float4_ShareMemory(M, K, N, d_A, d_B, d_C);

    // test_v8;
    // Double_Buffer_RMem_SMem(M,K,N,d_A,d_B,d_C);

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop)); // ȷ���¼����
    checkCudaErrors(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("mysgemm time: %.3f ms\n", elapsed_time);
    //-----------------------------------------------------------------------------------------
    // dev2host
    checkCudaErrors(hipMemcpy(h_C_gpu, d_C, C_mem_size, hipMemcpyDeviceToHost));

    // cublas_calc �����ж��ٶ�
    checkCudaErrors(hipEventRecord(start, 0)); // 0 Ĭ����

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                N, M, K,
                &alpha, d_B, N, d_A, K, &beta, d_C_cublas, N);

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop)); // ȷ���¼����
    checkCudaErrors(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("cublas time: %.3f ms\n", elapsed_time);

    checkCudaErrors(hipMemcpy(h_C_cublas, d_C_cublas, M * N * sizeof(float), hipMemcpyDeviceToHost));
    // cpu_calc �����ж�׼ȷ��
    // cpu_gemm(h_A,h_B,h_C_cpu,M,N,K);

    // compare res
    printf("mysgemm ans is ");
    compare_ans(h_C_cublas, h_C_gpu, M, N);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_C_cublas);
    free(h_A);
    free(h_B);
    free(h_C_cpu);
    free(h_C_gpu);
    free(h_C_cublas);
    hipblasDestroy(handle);
    return 0;
}

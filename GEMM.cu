/*
    A -- [M, K]
    B -- [K, N]
    C -- [M, N] = A * B
*/

#include <cstdio>
#include <cstdlib>
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define OFFSET(row, col, ld) ((row) * (ld) + (col))

#define checkCudaErrors(func)                                                      \
    {                                                                              \
        hipError_t e = (func);                                                    \
        if (e != hipSuccess)                                                      \
            printf("%s %d CUDA: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
    }

//------------------ naive_sgemm ------------------//
__global__ void
gemm_v1(int M, int K, int N, float *d_A, float *d_B, float *d_C)
{
    int A_ROW = M;
    int B_ROW = K;
    int C_ROW = M;
    float tmp = 0.0f;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    for (int i = 0; i < K; i++)
    {
        tmp += d_A[OFFSET(idy, i, A_ROW)] * d_B[OFFSET(i, idx, B_ROW)];
    }
    d_C[OFFSET(idy, idx, C_ROW)] = tmp;
}


//------------------ block_gemm ------------------//
template<unsigned int BLOCK_SIZE>
__global__ void
gemm_v2(int M, int K, int N, float *d_A, float *d_B, float *d_C)
{
    // 在 SMem 中存储 d_A 和 d_B 的块 读取
    __shared__ float A_block[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float B_block[BLOCK_SIZE][BLOCK_SIZE];

    // 找到这个线程的结果的存储在 d_C 的坐标
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    // 在分块矩阵中的坐标
    int tx = threadIdx.x, ty = threadIdx.y;

    float sum = 0.0f;
    for (int k = 0; k < K; k += BLOCK_SIZE)
    {
        // 把数据中从 GMem 中记录到 SMem 中

        A_block[ty][tx] = d_A[OFFSET(row, k + tx, K)]; // row * K + (k + tx)
        B_block[ty][tx] = d_B[OFFSET(k + ty, col, N)]; //(k + ty) * N + col
        // 同步 下一步要用共享内存的数据
        __syncthreads();

        for (int inner_k = 0; inner_k < BLOCK_SIZE; inner_k++)
            sum += A_block[ty][inner_k] * B_block[inner_k][tx];

        // 同步 下一步循环要清空SMem 必须要把数据用完
        __syncthreads();
    }

    d_C[OFFSET(row, col, N)] = sum; // row * N + col
}

//------------------ idel_gemm ------------------//
template<unsigned int BLOCK_SIZE>
__global__ void
gemm_v3(int M, int K, int N, float *d_A, float *d_B, float *d_C)
{
    // 在 SMem 中存储 d_A 和 d_B 的块 读取
    __shared__ float A_block[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float B_block[BLOCK_SIZE][BLOCK_SIZE];

    // 找到这个线程的结果的存储在 d_C 的坐标 1 2 3 4
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    // 在分块矩阵中的坐标
    int tx = threadIdx.x, ty = threadIdx.y;

    float sum1 = 0.0f, sum2 = 0.0f, sum3 = 0.0f, sum4 = 0.0f;
    for (int k = 0; k < K; k += BLOCK_SIZE)
    {
        // 把数据中从 GMem 中记录到 SMem 中
        A_block[ty][tx] = d_A[OFFSET(row, k + tx, K)]; // row * K + (k + tx)
        A_block[ty][tx+ BLOCK_SIZE / 2] = d_A[OFFSET(row, k + tx+ BLOCK_SIZE / 2, K)]; // row * K + (k + tx)
        A_block[ty+ BLOCK_SIZE / 2][tx] = d_A[OFFSET(row+ BLOCK_SIZE / 2, k + tx, K)]; // row * K + (k + tx)
        A_block[ty+ BLOCK_SIZE / 2][tx+ BLOCK_SIZE / 2] = d_A[OFFSET(row+ BLOCK_SIZE / 2, k + tx+ BLOCK_SIZE / 2, K)]; // row * K + (k + tx)
        B_block[ty][tx] = d_B[OFFSET(k + ty, col, N)]; //(k + ty) * N + col
        B_block[ty][tx+ BLOCK_SIZE / 2] = d_B[OFFSET(row, k + tx+ BLOCK_SIZE / 2, K)]; //(k + ty) * N + col
        B_block[ty+ BLOCK_SIZE / 2][tx] = d_B[OFFSET(row+ BLOCK_SIZE / 2, k + tx, K)]; //(k + ty) * N + col
        B_block[ty+ BLOCK_SIZE / 2][tx+ BLOCK_SIZE / 2] = d_B[OFFSET(row+ BLOCK_SIZE / 2, k + tx+ BLOCK_SIZE / 2, K)]; //(k + ty) * N + col
        // 同步 下一步要用共享内存的数据
        __syncthreads();

        for (int inner_k = 0; inner_k < BLOCK_SIZE / 2; inner_k++)
        {
            sum1 += A_block[ty][inner_k] * B_block[inner_k][tx];
            sum2 += A_block[ty][inner_k] * B_block[inner_k][tx + BLOCK_SIZE / 2];
            sum3 += A_block[ty][inner_k] * B_block[inner_k][tx + BLOCK_SIZE / 2];
            sum4 += A_block[ty + BLOCK_SIZE / 2][inner_k] * B_block[inner_k][tx + BLOCK_SIZE / 2];
        }
        // 同步 下一步循环要清空SMem 必须要把数据用完
        __syncthreads();
    }

    d_C[OFFSET(row, col, N)] = sum1; // row * N + col
    d_C[OFFSET(row, col + BLOCK_SIZE / 2 , N)] = sum2;
    d_C[OFFSET(row+ BLOCK_SIZE / 2, col, N)]= sum3;
    d_C[OFFSET(row+ BLOCK_SIZE / 2, col+ BLOCK_SIZE / 2, N)]= sum4;
}